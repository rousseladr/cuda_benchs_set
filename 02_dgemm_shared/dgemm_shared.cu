/*
 Copyright 2023 Adrien Roussel <adrien.roussel@protonmail.com>
 SPDX-License-Identifier: CECILL-C
*/

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#include <math.h>

#define BLOCK_WIDTH 32
#define TILE_WIDTH 32
#define TAILLE 4096


__global__
	void MulMatrixKernel(float* A, float* B, float* C, int N)
	{
		int col		= threadIdx.x + blockDim.x * blockIdx.x;
		int ligne	= threadIdx.y + blockDim.y * blockIdx.y;

		if((col < N) && (ligne < N))
    {
			float val = 0.0f;
			for(int k = 0; k < N; k++)
      {
				val += A[ligne * N + k] * B[k * N + col];
			}
			C[ligne * N + col] = val;
		}
	}

__global__
	void MulMatrixShare(float* A, float* B, float* C, int N){
		__shared__ float s_A[TILE_WIDTH][TILE_WIDTH];
		__shared__ float s_B[TILE_WIDTH][TILE_WIDTH];

		int ligne	= blockIdx.y * BLOCK_WIDTH + threadIdx.y;
		int col	  = blockIdx.x * BLOCK_WIDTH + threadIdx.x;

		float value = 0.0f;

		for(int id_tile = 0; id_tile < gridDim.x; id_tile++)
    {
      int i = id_tile * TILE_WIDTH + threadIdx.x;
      int j = id_tile * TILE_WIDTH + threadIdx.y;

			s_A[threadIdx.y][threadIdx.x] = A[ligne * N + j]; // charger un �l�ment de A [un �l�ment par thread]
			s_B[threadIdx.y][threadIdx.x] = B[i * N + col]; // charger un �l�ment de B [un �l�ment par thread]

      // Attente que tous les threads ont bien charg� dans la m�moire partag�e leurs deux indices
			__syncthreads();

			for(int k =0; k < TILE_WIDTH; k++)
      {
				value += s_A[threadIdx.y][k] * s_B[k][threadIdx.x];
      }

      // S'assurer que tous les threads ont bien fini le calcul du pr�liminaire du tile courant avant de commencer la prochaine �tape du calcul de cette tile
			__syncthreads();
		}

    // Enregistrer la valeur accumul�e dans C (m�moire globale)
		C[ligne * N + col] = value;
}

int main(int argc, char** argv)
{
	int N = (argc >= 2)?(atoi(argv[1])):TAILLE;
	int nbBlocks = N / BLOCK_WIDTH;
	//if(N % BLOCK_WIDTH) nbBlocks++;
	if(N % BLOCK_WIDTH) N += (N % BLOCK_WIDTH);
	dim3 gridSize(nbBlocks, nbBlocks);
	dim3 blockSize(BLOCK_WIDTH, BLOCK_WIDTH);

	float *A, *B, *C;
	float *d_A, *d_B, *d_C;

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	A = (float*) malloc(sizeof(float) * N * N);
	B = (float*) malloc(sizeof(float) * N * N);
	C = (float*) malloc(sizeof(float) * N * N);

	hipMalloc(&d_A, sizeof(float) * N * N);
	hipMalloc(&d_B, sizeof(float) * N * N);
	hipMalloc(&d_C, sizeof(float) * N * N);

	srand(2019);

	for(int i = 0; i < N * N; i++)
  {
		A[i] = rand();
		B[i] = rand();
		C[i] = 0.0f;
	}

	hipMemcpy(d_A, A, sizeof(float) * N * N, hipMemcpyHostToDevice);
	hipMemcpy(d_B, B, sizeof(float) * N * N, hipMemcpyHostToDevice);
	hipMemcpy(d_C, C, sizeof(float) * N * N, hipMemcpyHostToDevice);

	hipEventRecord(start);
	//MulMatrixKernel<<<gridSize, blockSize>>>(d_A, d_B, d_C, N);
	MulMatrixShare<<<gridSize, blockSize>>>(d_A, d_B, d_C, N);
	hipEventRecord(stop);

	hipMemcpy(C, d_C, sizeof(float) * N * N, hipMemcpyDeviceToHost);

	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("Matrice %dx%d\n\tTemps: %f s\n", N, N, milliseconds/1000);
	//printf("%f", milliseconds/1000);

	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);

	free(A);
	free(B);
	free(C);

	return 0;
}
